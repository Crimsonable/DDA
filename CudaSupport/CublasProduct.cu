#include "hip/hip_runtime.h"
#include "CublasProduct.cuh"

__global__ static void show(float* C, size_t pitch, int r, int c) {
	for (int i = 0; i < r; ++i) {
		for (int j = 0; j < c; ++j) {
			printf("%f ", *((float*)((char*)C + sizeof(float)*i + j * pitch)));
		}
		printf("\n");
	}
}


extern "C" void cudaProductS(float* A, float* B, float* C, int m, int n, int k) {
	float alpha = 1.0, beta = 0.0;
	float* d_A, *d_B, *d_C;
	size_t size_A = sizeof(float)*m*k, size_B = sizeof(float)*k*n, size_C = sizeof(float)*m*n;
	/*std::size_t pitchA, pitchB, pitchC;
	hipMallocPitch(&d_A, &pitchA, m * sizeof(float), n);
	hipMallocPitch(&d_B, &pitchB, k * sizeof(float), n);
	hipMallocPitch(&d_C, &pitchC, m * sizeof(float), n);

	hipError_t error = hipMemcpy2D(d_A, pitchA, A, sizeof(float)*m, sizeof(float)*m, k, hipMemcpyHostToDevice);
	hipError_t error2 = hipMemcpy2D(d_B, pitchB, B, sizeof(float)*k, sizeof(float)*k, n, hipMemcpyHostToDevice);*/
	hipMalloc(&d_A, size_A);
	hipMalloc(&d_B, size_B);
	hipMalloc(&d_C, size_C);

	hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice);

	/*dim3 grid(1, 1);
	dim3 block(1, 1);
	show << <grid, block >> > (d_A, sizeof(float)*m, m, k);
	show << <grid, block >> > (d_B, sizeof(float)*k, k, n);*/

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_A, m, d_B, k, &beta, d_C, m);
	//show << <grid, block >> > (d_C, sizeof(float)*m, m, n);

	//hipMemcpy2D(C, m * sizeof(float), d_C, pitchC, sizeof(float)*m, n, hipMemcpyDeviceToHost);
	hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}

extern "C" void cudaProductD(double* A, double* B, double* C, int m, int n, int k) {
	double alpha = 1.0, beta = 0.0;
	double* d_A, *d_B, *d_C;
	size_t size_A = sizeof(double)*m*k, size_B = sizeof(double)*k*n, size_C = sizeof(double)*m*n;
	/*std::size_t pitchA, pitchB, pitchC;
	hipMallocPitch(&d_A, &pitchA, m * sizeof(float), n);
	hipMallocPitch(&d_B, &pitchB, k * sizeof(float), n);
	hipMallocPitch(&d_C, &pitchC, m * sizeof(float), n);

	hipError_t error = hipMemcpy2D(d_A, pitchA, A, sizeof(float)*m, sizeof(float)*m, k, hipMemcpyHostToDevice);
	hipError_t error2 = hipMemcpy2D(d_B, pitchB, B, sizeof(float)*k, sizeof(float)*k, n, hipMemcpyHostToDevice);*/
	hipMalloc(&d_A, size_A);
	hipMalloc(&d_B, size_B);
	hipMalloc(&d_C, size_C);

	hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice);

	/*dim3 grid(1, 1);
	dim3 block(1, 1);
	show << <grid, block >> > (d_A, sizeof(float)*m, m, k);
	show << <grid, block >> > (d_B, sizeof(float)*k, k, n);*/

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_A, m, d_B, k, &beta, d_C, m);
	//show << <grid, block >> > (d_C, sizeof(float)*m, m, n);

	//hipMemcpy2D(C, m * sizeof(float), d_C, pitchC, sizeof(float)*m, n, hipMemcpyDeviceToHost);
	hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}
